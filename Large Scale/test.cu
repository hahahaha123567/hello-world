#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

const int userNum = 1024;
const int itemNum = 2048;
const int infoNum = 80000;
const int requestNum = 20000;

// <<< 2048 >>>
__global__ void calcuLength(int **d_trainMatrix, float * lenVector)
{
	int item = blockIdx.x;
	int sum = 0;
	for (int i = 0; i < userNum; ++i) {
		if (d_trainMatrix[i][item] == 0) {
			continue;
		}
		sum += d_trainMatrix[i][item] * d_trainMatrix[i][item];
	}
	lenVector[item] = sqrt((float)sum);
	__syncthreads();
}

// <<< 4096, 1024>>>
__global__ void calcuSim(int ** d_trainMatrix, float * lenVector, float ** d_sim)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int item1 = index / 1024;
	int item2 = index % 1024;
	int sum = 0;
	for (int i = 0; i < userNum; ++i){
		sum += d_trainMatrix[i][item1] * d_trainMatrix[i][item2];
	}
	if (lenVector[item1] && lenVector[item2])
		d_sim[item1][item2] = d_sim[item2][item1] = sum/lenVector[item1]/lenVector[item2];
	else 
		d_sim[item1][item2] = d_sim[item2][item1] = 0;
	__syncthreads();
	// if (item1 < 100 && item2 < 100)
	// printf("%d and %d: %f\n", item1, item2, d_sim[item1][item2]);
}

// <<<32, 1024>>>
__global__ void calcuGrade(int ** d_trainMatrix, float ** d_sim, int * userVec, int * itemVec) 
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	printf("%d\n", index);return;
	if (index >= requestNum) return;
	int userID = userVec[index];
	int itemID = itemVec[index];

	float sum = 0;
	int count = 0;
	for (int i = 0; i < itemNum; ++i) {
		if (d_trainMatrix[userID][i]) {
			sum += d_sim[i][itemID] * d_trainMatrix[userID][i];
			count++;
		}
	}
	d_trainMatrix[userID][itemID] = sum / count;
	printf("%f\n", d_trainMatrix[userID][itemID]);
}

__global__ void show(float * fuck)
{
	printf("%f\n", fuck[10000]);
}

int main()
{	hipError_t error;
	// input .base to h_trainMatrix
	int ** h_trainMatrix, * tempSpace;
	h_trainMatrix = (int **)malloc(userNum * sizeof(int *));
	tempSpace = (int *)malloc(itemNum * userNum * sizeof(int));
	for (int i = 0, j = 0; i < userNum; ++i, j+=itemNum) {
		h_trainMatrix[i] = &tempSpace[j];
	}
	FILE * fp = fopen("u1.base", "r");
	int userID, itemID, grade, time;
	for (int i = 0; i < infoNum; ++i) {
		fscanf(fp, "%d %d %d %d", &userID, &itemID, &grade, &time);
		h_trainMatrix[userID][itemID] = grade;
	}
	fclose(fp);

	// copy h_trainMatrix to d_trainMatrix
	int ** d_trainMatrix = NULL;	
	hipMalloc((void **)&d_trainMatrix, userNum * sizeof(int *));
	hipMalloc((void **)&tempSpace, itemNum * userNum * sizeof(int));
	int ** temptemp = (int **)malloc(userNum * sizeof(int *));
	for (int i = 0, j = 0; i < userNum; ++i, j+=itemNum) {
		temptemp[i] = &tempSpace[j];
	}
	hipMemcpy(d_trainMatrix, temptemp, userNum * sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(tempSpace, *h_trainMatrix, userNum * itemNum * sizeof(int), hipMemcpyHostToDevice);
	
	// calculate length
	float * lenVector = NULL;
	hipMalloc((void **)&lenVector, itemNum * sizeof(float));
	calcuLength <<<2048, 1>>> (d_trainMatrix, lenVector);

	// calculate similarity
	float ** d_sim = NULL;
	hipMalloc((void **)&d_sim, itemNum * sizeof(float *));
	float ** temptemp1 = (float **)malloc(itemNum * sizeof(float *));
	float * tempSpace1;
	hipMalloc((void **)&tempSpace1, itemNum * itemNum * sizeof(float));
	for (int i = 0, j = 0; i < itemNum; ++i, j+=itemNum) {
		temptemp1[i] = &tempSpace1[j];
	}
	hipMemcpy(d_sim, temptemp1, itemNum * sizeof(float *), hipMemcpyHostToDevice);
	calcuSim <<<4096, 1024>>> (d_trainMatrix, lenVector, d_sim);

	// // copy similarity to host
	// float ** h_sim = (float **)malloc(itemNum * sizeof(float *));
	// float * tempSpace2 = (float *)malloc(itemNum * itemNum * sizeof(float));
	// !!! unknown error hipMemcpy(tempSpace2, tempSpace1, itemNum * itemNum * sizeof(float), hipMemcpyDeviceToHost);	
	// for (int i = 0, j = 0; i < itemNum; ++i, j+=itemNum) {
	// 	h_sim[i] = &tempSpace2[j];
	// }
	
	// input .test and calculate
	fp = fopen("u1.test", "r");
	int temp1, temp2;
	int * h_userVec = (int *)malloc(requestNum * sizeof(int));
	int * h_itemVec = (int *)malloc(requestNum * sizeof(int));
	int * d_userVec, * d_itemVec;
	hipMalloc((void **)&d_userVec, requestNum * sizeof(int));
	hipMalloc((void **)&d_itemVec, requestNum * sizeof(int));
	for (int i = 0; i < requestNum; ++i) {
		fscanf(fp, "%d %d %d %d", h_userVec+i, h_itemVec+i, &temp1, &temp2);
	}
	fclose(fp);
error = hipGetLastError();
printf("CUDA error: %s\n", hipGetErrorString(error));
	hipMemcpy(d_userVec, h_userVec, requestNum * sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpy(d_itemVec, h_itemVec, requestNum * sizeof(int), hipMemcpyHostToDevice);	
error = hipGetLastError();
printf("CUDA error: %s\n", hipGetErrorString(error));
	calcuGrade <<<1, 1>>> (d_trainMatrix, d_sim, d_userVec, d_itemVec);

	fp = fopen("u1.ans", "w");

	fclose(fp);

	// free
	hipFree(d_trainMatrix);
	free(h_trainMatrix);
}
